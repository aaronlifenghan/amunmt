#include "hip/hip_runtime.h"
#include "matrix.h"

namespace GPU {

namespace mblas {

#ifdef __APPLE__
boost::thread_specific_ptr<hipblasHandle_t> CublasHandler::handle_;
#else
thread_local hipblasHandle_t* CublasHandler::handle_ = nullptr;
#endif

Matrix& Swap(Matrix& Out, Matrix& In) {
  size_t iRows = In.Rows();
  size_t iCols = In.Cols();
  size_t oRows = Out.Rows();
  size_t oCols = Out.Cols();

  Out.Reshape(iRows, iCols);
  In.Reshape(oRows, oCols);

  In.GetVec().swap(Out.GetVec());
  return Out;
}

Matrix& Mean(Matrix& Out, const Matrix& In) {
  size_t m = In.Rows();
  size_t n = In.Cols();

  Out.Resize(1, n, 0.f);
  Matrix Ones(1, m, 1.f);

  float alpha = 1.0 / m;
  float beta  = 0.0;
  hipblasSgemv(CublasHandler::GetHandle(), HIPBLAS_OP_N, n, m, &alpha, In.data(), n,
              Ones.data(), 1, &beta, Out.data(), 1);
  return Out;
}

Matrix& Transpose(Matrix& Out, const Matrix& In) {
  size_t m = In.Rows();
  size_t n = In.Cols();

  Out.Resize(n, m);

  float alpha = 1.0;
  float beta  = 0.0;

  hipblasSgeam(CublasHandler::GetHandle(), HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, In.data(), n,
              &beta, In.data(), n, Out.data(), m);

  return Out;
}

Matrix& Transpose(Matrix& Out) {
  Matrix Temp;
  Transpose(Temp, Out);
  Swap(Out, Temp);
  return Out;
}

Matrix& Concat(Matrix& Out, const Matrix& In) {
  size_t oldSize = Out.size();
  Out.Resize(Out.Rows() + In.Rows(), Out.Cols());
  lib::copy(In.begin(), In.end(), Out.begin() + oldSize);
  return Out;
}

Matrix& Copy(Matrix& Out, const Matrix& In) {
  Out.Resize(In.Rows(), In.Cols());
  lib::copy(In.begin(), In.end(), Out.begin());
  return Out;
}

Matrix& PasteRow(Matrix& Out,
                 const Matrix& In,
                 const size_t r, const size_t c) {
  size_t start = r * Out.Cols() + c;
  lib::copy(In.begin(), In.end(), Out.begin() + start);
  return Out;
}

Matrix& CopyRow(Matrix& Out,
                const Matrix& In,
                const size_t r, const size_t c) {
  size_t length = In.Cols() - c;
  Out.Resize(1, length);
  size_t start = r * In.Cols() + c;
  size_t end   = start + length;
  lib::copy(In.begin() + start, In.begin() + end, Out.begin());
  return Out;
}

__global__ void gCopyRows(float* out, const float* in, size_t cols,
                          const RowPair* devPairs, size_t numPairs) {
  for(int bid = 0; bid < numPairs; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < numPairs) {
      size_t dstId = devPairs[j].first;
      size_t srcId = devPairs[j].second;

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

Matrix& CopyRows(Matrix& Out,
                 const Matrix& In,
                 const RowPair* devPairs,
                 size_t numPairs) {
  float* d_out = Out.data();
  const float* d_in = In.data();

  int threads = std::min(MAX_THREADS, (int)In.Cols());
  int blocks = std::min(MAX_BLOCKS, (int)numPairs);;
  gCopyRows<<<blocks, threads>>>(d_out, d_in, In.Cols(), devPairs, numPairs);
  hipStreamSynchronize(0);
  return Out;
}

Matrix& CopyRows(Matrix& Out,
                 const Matrix& In,
                 const RowPairs& pairs) {
  thrust::device_vector<RowPair> devPairs = pairs;
  CopyRows(Out, In, thrust::raw_pointer_cast(devPairs.data()), devPairs.size());
  return Out;
}

Matrix& Assemble(Matrix& Out,
                 const Matrix& In,
                 const std::vector<size_t>& indeces) {
  RowPairs rowPairs;
  for(size_t i = 0; i < indeces.size(); i++)
    rowPairs.emplace_back(i, indeces[i]);
  Out.Resize(rowPairs.size(), In.Cols());
  CopyRows(Out, In, rowPairs);
  return Out;
}

__global__ void gSlice(float* out, const float* in,
                       size_t n, size_t dim,
                       size_t rows, size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * dim;
      const float* rowIn = in + j * cols + n * dim;

      for(int tid = 0; tid < dim; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < dim)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

Matrix& Slice(Matrix& Out,
              const Matrix& In,
              size_t n, size_t dim) {

  Out.Resize(In.Rows(), dim);

  float* d_out = Out.data();
  const float* d_in = In.data();

  int threads = std::min(MAX_THREADS, (int)dim);
  int blocks = std::min(MAX_BLOCKS, (int)In.Rows());
  gSlice<<<blocks, threads>>>(d_out, d_in, n, dim, In.Rows(), In.Cols());
  hipStreamSynchronize(0);
  return Out;
}

Matrix& Prod(hipblasHandle_t handle, Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {
  Matrix::value_type alpha = 1.0;
  Matrix::value_type beta = 0.0;

  //size_t m = A.Rows();
  //size_t k = A.Cols();
  ////if(transA)
  ////  std::swap(m, k);
  //
  //size_t l = B.Rows();
  //size_t n = B.Cols();
  ////if(transB)
  ////  std::swap(l, n);
  //
  //C.Resize(m, n);
  //
  //size_t lda = A.Cols();
  //size_t ldb = B.Cols();
  //size_t ldc = C.Cols();
  //
  //nervana_sgemm(const_cast<float*>(A.data()),
  //              const_cast<float*>(B.data()),
  //              C.data(),
  //              transA, transB,
  //              m, n, k,
  //              lda, ldb, ldc,
  //              alpha, beta,
  //              0, false, false, 0);

  size_t m = A.Rows();
  size_t k = A.Cols();
  if(transA)
    std::swap(m, k);

  size_t l = B.Rows();
  size_t n = B.Cols();
  if(transB)
    std::swap(l, n);

  size_t lda = A.Cols();
  size_t ldb = B.Cols();
  size_t ldc = B.Cols();

  if(transB)
    ldc = B.Rows();

  C.Resize(m, n);

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  hipblasSgemm(handle, opB, opA,
              n, m, k, &alpha, B.data(), ldb, A.data(), lda, &beta, C.data(), ldc);
  return C;
}

Matrix& Prod(Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {

 return Prod(CublasHandler::GetHandle(), C, A, B, transA, transB);
}

__global__ void gSoftMax(float* softMaxP, size_t rows, size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;
      float* sp = softMaxP + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          sp[id] = __expf(sp[id]);
          _sum[threadIdx.x] += sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x){
        int id = tid + threadIdx.x;
        if(id < cols)
          sp[id] /= _sum[0];
      }
    }
  }
}

Matrix& Softmax(Matrix& Out) {
  int blocks = std::min(MAX_BLOCKS, (int)Out.Rows());
  int threads = std::min(MAX_THREADS, (int)Out.Cols());
  int shared = sizeof(float) * threads * 2;
  gSoftMax<<<blocks, threads, shared>>>(Out.data(), Out.Rows(), Out.Cols());
  hipStreamSynchronize(0);
  return Out;
}

}

}

